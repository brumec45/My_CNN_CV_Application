#include "hip/hip_runtime.h"
#include "Output_Layer_GPU_Kernels.cuh"

__constant__ float anchors_416[10] = { 1.08, 1.19,  3.42, 4.41,  6.63, 11.38,  9.42, 5.11,  16.62, 10.52 };

__device__ float Sigmoid(float x) 
{
	float expValue = exp((double)-x);
	float result = 1 / (1 + expValue);

	return result;
}

__global__ void XY_BoundingBox_Coordinates_Transform_Kernel(float* input, int inputHeight, int inputWidth) 
{
	int threadIndex = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	int tensorXYSize = inputHeight * inputWidth;
	int tensorSize = boundingBoxesPerGridCell * tensorXYSize;
	

	if (threadIndex < tensorSize)
	{
		int threadDepthIndex = threadIndex % boundingBoxesPerGridCell;
		//int threadDepthIndexY = (threadIndex % XYCoordinatesCount) + 1;
		int threadXYIndex = threadIndex % tensorXYSize;
		int cy = threadXYIndex / inputWidth;
		int cx = threadXYIndex % inputWidth;
				
		//tensor[threadDepthIndex * tensorXYSize + threadXYIndex] = threadDepthIndex;
		input[threadDepthIndex * 4 * tensorXYSize + threadXYIndex] = (cx + Sigmoid(input[threadDepthIndex * 4 * tensorXYSize + threadXYIndex])) * downsampleFactor;
		input[(threadDepthIndex * 4 + 1) * tensorXYSize + threadXYIndex] = (cy + Sigmoid(input[(threadDepthIndex * 4 + 1) * tensorXYSize + threadXYIndex])) * downsampleFactor;
		//input[threadDepthIndex * 4 * tensorXYSize + threadXYIndex] = 1;
		//input[(threadDepthIndex * 4 + 1) * tensorXYSize + threadXYIndex] = 1;
	}
}
__global__ void WH_BoundingBox_Transform_Kernel(float* input, int inputHeight, int inputWidth) 
{
	int threadIndex = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	int tensorXYSize = inputHeight * inputWidth;
	int tensorSize = boundingBoxesPerGridCell * tensorXYSize;

	if (threadIndex < tensorSize)
	{
		int threadDepthIndex = threadIndex % boundingBoxesPerGridCell;
		//int threadDepthIndexY = (threadIndex % XYCoordinatesCount) + 1;
		int threadXYIndex = threadIndex % tensorXYSize;
		
		//tensor[threadDepthIndex * tensorXYSize + threadXYIndex] = threadDepthIndex;
		input[(threadDepthIndex * 4 + 2) * tensorXYSize + threadXYIndex] = exp(input[(threadDepthIndex * 4 + 2) * tensorXYSize + threadXYIndex]) *
			anchors_416[2 * threadDepthIndex] * downsampleFactor;
		input[(threadDepthIndex * 4 + 3) * tensorXYSize + threadXYIndex] = exp(input[(threadDepthIndex * 4 + 3) * tensorXYSize + threadXYIndex]) *
			anchors_416[2 * threadDepthIndex + 1] * downsampleFactor;
		//input[(threadDepthIndex * 4 + 2) * tensorXYSize + threadXYIndex] = anchors_416[2 * threadDepthIndex] = 1;
		//input[(threadDepthIndex * 4 + 3) * tensorXYSize + threadXYIndex] = anchors_416[2 * threadDepthIndex + 1] = 1;

		input[(20 + threadDepthIndex) * tensorXYSize + threadXYIndex] = Sigmoid(input[(20 + threadDepthIndex) * tensorXYSize + threadXYIndex]);
		//input[(20 + threadDepthIndex) * tensorXYSize + threadXYIndex] = 2;
	}
}

__global__ void Softmax_Kernel(float* input, int classesCount, int inputHeight, int inputWidth)
{
	int threadIndex = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	int tensorXYSize = inputHeight * inputWidth;
	int tensorSize = boundingBoxesPerGridCell * tensorXYSize;

	if (threadIndex < tensorSize)
	{
		int threadDepthIndex = threadIndex % boundingBoxesPerGridCell;
		int threadXYIndex = threadIndex % tensorXYSize;
		float maxClassProbability = FLOAT_MIN;

		for (size_t i = 0; i < classesCount; i++)
		{
			float classProbability = input[(25 + threadDepthIndex * classesCount + i) * tensorXYSize + threadXYIndex];

			if (classProbability > maxClassProbability)
			{
				maxClassProbability = classProbability;
			}
		}

		float classProbabilitiesSum = 0;
		for (size_t i = 0; i < classesCount; i++)
		{
			float exponent = exp(input[(25 + threadDepthIndex * classesCount + i) * tensorXYSize + threadXYIndex] - maxClassProbability);
			classProbabilitiesSum += exponent;
			input[(25 + threadDepthIndex * classesCount + i) * tensorXYSize + threadXYIndex] = exponent;
		}

		for (size_t i = 0; i < classesCount; i++)
		{
			input[(25 + threadDepthIndex * classesCount + i) * tensorXYSize + threadXYIndex] /= classProbabilitiesSum;
			//input[(25 + threadDepthIndex * classesCount + i) * tensorXYSize + threadXYIndex] = i;
			//input[(25 + threadDepthIndex * classesCount + i) * tensorXYSize + threadXYIndex] = 3;
		}
	}
}


void WH_BoundingBox_Transform(float* input, int inputHeight, int inputWidth) 
{
	int tensorSize = boundingBoxesPerGridCell * inputHeight * inputWidth;
	int gridXDim = ceil(tensorSize / 512.0);
	WH_BoundingBox_Transform_Kernel << <gridXDim, 512 >> > (input, inputHeight, inputWidth);
}

void XY_BoundingBox_Coordinates_Transform(float* input, int inputHeight, int inputWidth)
{
	int tensorSize = boundingBoxesPerGridCell * inputHeight * inputWidth;
	int gridXDim = ceil(tensorSize / 512.0);
	XY_BoundingBox_Coordinates_Transform_Kernel << <gridXDim, 512 >> > (input, inputHeight, inputWidth);
}

void Softmax_GPU(float* input, int classesCount, int inputHeight, int inputWidth)
{
	int tensorSize = boundingBoxesPerGridCell * inputHeight * inputWidth;
	int gridXDim = ceil(tensorSize / 512.0);
	Softmax_Kernel << <gridXDim, 512 >> > (input, classesCount, inputHeight, inputWidth);
}