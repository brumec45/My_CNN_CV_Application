#include "hip/hip_runtime.h"
#include "Output_Layer_GPU_Kernels.cuh"

__constant__ float anchors_416[10] = { 1.08, 1.19,  3.42, 4.41,  6.63, 11.38,  9.42, 5.11,  16.62, 10.52 };

__device__ float Sigmoid(float x) 
{
	float expValue = exp((double)-x);
	float result = 1 / (1 + expValue);

	return result;
}

__global__ void XY_BoundingBox_Coordinates_Transform_Kernel(float* input, int inputHeight, int inputWidth) 
{
	int threadIndex = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	int tensorXYSize = inputHeight * inputWidth;
	int tensorSize = XYCoordinatesCount * tensorXYSize;
	

	if (threadIndex < tensorSize)
	{
		int threadDepthIndex = threadIndex % XYCoordinatesCount;
		//int threadDepthIndexY = (threadIndex % XYCoordinatesCount) + 1;
		int threadXYIndex = threadIndex % tensorXYSize;
		int cy = threadXYIndex / inputWidth;
		int cx = threadXYIndex % inputWidth;
				
		//tensor[threadDepthIndex * tensorXYSize + threadXYIndex] = threadDepthIndex;
		input[threadDepthIndex * 4 * tensorXYSize + threadXYIndex] = (cx + Sigmoid(input[threadDepthIndex * 4 * tensorXYSize + threadXYIndex])) * downsampleFactor;
		input[(threadDepthIndex * 4 + 1) * tensorXYSize + threadXYIndex] = (cy + Sigmoid(input[(threadDepthIndex * 4 + 1) * tensorXYSize + threadXYIndex])) * downsampleFactor;
		//input[threadDepthIndex * 4 * tensorXYSize + threadXYIndex] = threadDepthIndex * 4 * tensorXYSize + threadXYIndex;
		//input[(threadDepthIndex * 4 + 1) * tensorXYSize + threadXYIndex] = (threadDepthIndex * 4 + 1) * tensorXYSize + threadXYIndex;
		//input[threadDepthIndex * 4 * tensorXYSize + threadXYIndex] = cx;
		//input[(threadDepthIndex * 4 + 1) * tensorXYSize + threadXYIndex] = cy;
	}
}
__global__ void WH_BoundingBox_Transform_Kernel(float* input, int inputHeight, int inputWidth) 
{
	int threadIndex = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	int tensorXYSize = inputHeight * inputWidth;
	int tensorSize = WHCoordinatesCount * tensorXYSize;

	if (threadIndex < tensorSize)
	{
		int threadDepthIndex = threadIndex % XYCoordinatesCount;
		//int threadDepthIndexY = (threadIndex % XYCoordinatesCount) + 1;
		int threadXYIndex = threadIndex % tensorXYSize;
		int cy = threadXYIndex / inputWidth;
		int cx = threadXYIndex % inputWidth;

		//tensor[threadDepthIndex * tensorXYSize + threadXYIndex] = threadDepthIndex;
		input[(threadDepthIndex * 4 + 2) * tensorXYSize + threadXYIndex] = exp(input[(threadDepthIndex * 4 + 2) * tensorXYSize + threadXYIndex]) *
			anchors_416[2 * threadDepthIndex] * downsampleFactor;
		input[(threadDepthIndex * 4 + 3) * tensorXYSize + threadXYIndex] = exp(input[(threadDepthIndex * 4 + 3) * tensorXYSize + threadXYIndex]) *
			anchors_416[2 * threadDepthIndex + 1] * downsampleFactor;
		//input[(threadDepthIndex * 4 + 2) * tensorXYSize + threadXYIndex] = anchors_416[2 * threadDepthIndex];
		//input[(threadDepthIndex * 4 + 3) * tensorXYSize + threadXYIndex] = anchors_416[2 * threadDepthIndex + 1];
	}

}


void WH_BoundingBox_Transform(float* input, int inputHeight, int inputWidth) 
{
	int WHCoordinatesCount = 5;
	int tensorSize = WHCoordinatesCount * inputHeight * inputWidth;
	int gridXDim = ceil(tensorSize / 512.0);
	WH_BoundingBox_Transform_Kernel << <gridXDim, 512 >> > (input, inputHeight, inputWidth);
}

void Output_Transform_GPU(float* input, int inputHeight, int inputWidth)
{
	int XYCoordinatesCount = 5;
	int tensorSize = XYCoordinatesCount * inputHeight * inputWidth;
	int gridXDim = ceil(tensorSize / 512.0);
	XY_BoundingBox_Coordinates_Transform_Kernel << <gridXDim, 512 >> > (input, inputHeight, inputWidth);
}