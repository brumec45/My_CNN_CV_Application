#include "hip/hip_runtime.h"
#include "Leaky_RELU_Activation_Kernel.cuh"

__global__ void Leaky_RELU_Kernel(float* tensor, int tensorSize)
{
	int threadIndex = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (threadIndex < tensorSize)
	{
		if (tensor[threadIndex] > 0)
		{
			tensor[threadIndex] = tensor[threadIndex];
		}
		else
		{
			tensor[threadIndex] = tensor[threadIndex] * 0.1f;
		}
	}
}

void LeakyRELUActivation(float* tensor, int tensorSize) {
	//vedno je deljivo z 512 (4)
	Leaky_RELU_Kernel << <tensorSize / 512, 512 >> > (tensor, tensorSize);
}