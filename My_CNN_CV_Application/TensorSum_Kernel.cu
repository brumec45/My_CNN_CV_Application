#include "hip/hip_runtime.h"
#include "TensorSum_Kernel.cuh"


__global__ void AddBIAS_Kernel(float* tensor, float* bias, int biasSize, int tensorHeight, int tensorWidth)
{
	int threadIndex = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	int tensorXYSize = tensorHeight * tensorWidth;
	int tensorSize = biasSize * tensorXYSize;

	if (threadIndex < tensorSize)
	{
		int threadDepthIndex = threadIndex % biasSize;
		int threadXYIndex = threadIndex % tensorXYSize;

		//tensor[threadDepthIndex * tensorXYSize + threadXYIndex] = threadDepthIndex;
		tensor[threadDepthIndex * tensorXYSize + threadXYIndex] += bias[threadDepthIndex];
	}
}


void AddBIAS_GPU(float* tensor, float* bias, int biasSize, int tensorHeight, int tensorWidth) {
	
	int tensorSize = biasSize * tensorHeight * tensorWidth;
	int gridXDim = ceil(tensorSize / 512.0);
	AddBIAS_Kernel << <gridXDim, 512 >> > (tensor, bias, biasSize, tensorHeight, tensorWidth);
}

