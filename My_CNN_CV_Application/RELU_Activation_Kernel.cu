#include "hip/hip_runtime.h"
#include "RELU_Activation_Kernel.cuh"

__global__ void Leaky_RELU_Kernel(float* tensor, int tensorSize)
{
	int threadIndex = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (threadIndex < tensorSize)
	{
		if (tensor[threadIndex] > 0)
		{
			tensor[threadIndex] = tensor[threadIndex];
		}
		else
		{
			tensor[threadIndex] = tensor[threadIndex] * 0.1f;
		}
	}
}

__global__ void RELU_Kernel(float* tensor, int tensorSize)
{
	int threadIndex = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (threadIndex < tensorSize)
	{
		tensor[threadIndex] = tensor[threadIndex] * (tensor[threadIndex] > 0);
	}
}

void LeakyRELUActivation(float* tensor, int tensorSize) {
	//vedno je deljivo z 512 (4)
	Leaky_RELU_Kernel << <tensorSize / 512, 512 >> > (tensor, tensorSize);
}

void RELUActivation(float* tensor, int tensorSize) {
	//vedno je deljivo z 512 (4)
	RELU_Kernel << <tensorSize / 512, 512 >> > (tensor, tensorSize);
}